#include "hip/hip_runtime.h"
//
//  CanonicityChecker.cu
//  BIBD_Mac
//
//  Created by Andrei Ivanov on 2/10/18.
//  Copyright (c) 2018 Andrei Ivanov. All rights reserved.
//

#include "CanonicityChecker.cpp"

#if !CONSTR_ON_GPU
GroupOrderInfo *CGPU_CheckerInfo<MATRIX_ELEMENT_TYPE>::m_pOrderInfo;
size_t CGPU_CheckerInfo<MATRIX_ELEMENT_TYPE>::m_nCPUthreads;
COrderInfo **CGPU_CheckerInfo<MATRIX_ELEMENT_TYPE>::m_ppOrderInfo;
CTimerInfo CGPU_CheckerInfo<MATRIX_ELEMENT_TYPE>::m_timer;

template<class T>
__global__ void AssignCheckerGlobal(CMatrixCanonCheckerGPU<T> **ppCheckers, uint checkerIdx, CMatrixData<T> *pMatrix, 
	T nRows, T nCols, T maxElem, bool IS_enum, const size_t *pColOrbInfo)
{
	auto pChecker = ppCheckers[checkerIdx];
	if (!pChecker) {
#if USE_OLD_CODE
		pChecker = ppCheckers[checkerIdx] = new CMatrixCanonChecker<T>(pMatrix, nRows, nCols, maxElem, IS_enum);
#else
		pMatrix->InitWithData(nRows, nCols, maxElem);
		pChecker = ppCheckers[checkerIdx] = new CMatrixCanonCheckerGPU<T>(pMatrix);
		pChecker->initiateColOrbits(nRows, IS_enum, NULL);
		if (!checkerIdx)
			pChecker->setEnumInfo(new CInsSysEnumInfo<T>());
#endif
	}

	// Restore ColOrbit information on GPU
	pChecker->restoreColOrbitInfo(nRows, pColOrbInfo);
}

bool AssignChecker(CMatrixCanonCheckerGPU<T, S> **ppCheckers, CMatrixData<T, S> **pAllMatrData, uint checkerIdx,
					const CEnumerator<T, S> *pCanonChecker, hipStream_t stream
#if TRACE_CUDA_FLAG
					, int myID
#endif
					)
{
	const CMatrixData<MATRIX_ELEMENT_TYPE> *pMatrix = pCanonChecker->matrix();
	auto nRows = pMatrix->rowNumb();

	// Converting ColOrbit information to GPU
	const size_t nColOrb = pCanonChecker->copyColOrbitInfo(nRows);

	const auto pColOrbInfoBeg = pCanonChecker->GPU_CanonChecker()->ColOrbitData(t_CPU);
	auto pColOrbitDataCPU = pCanonChecker->GPU_CanonChecker()->ColOrbitData(t_GPU);
	CudaSafeCall(hipMemcpyAsync(pColOrbitDataCPU, pColOrbInfoBeg,
		nColOrb * sizeof(pColOrbInfoBeg[0]), hipMemcpyHostToDevice, stream));

	// Copying the matrix
	const hipError_t err = hipMemcpyAsync((char *)pAllMatrData[checkerIdx] + sizeof(CMatrixData<MATRIX_ELEMENT_TYPE>),
								pMatrix->GetDataPntr(), pMatrix->lenData(), hipMemcpyHostToDevice, stream);
	TRACE_CUDA("  assignChecker 3: pAllMatrData[checkerIdx = %d] = %p  err = %d%s\n", checkerIdx, pAllMatrData[checkerIdx], err, err != hipSuccess ? " - ERROR" : "")
	CudaSafeCallRet(err, false);

	AssignCheckerGlobal <<<1, 1, 0, stream>>> (ppCheckers, checkerIdx, pAllMatrData[checkerIdx],
									nRows, pMatrix->colNumb(), pMatrix->maxElement(), 
									pCanonChecker->IS_enumerator(), pColOrbitDataCPU);
	TRACE_CUDA("  assignChecker 4: err = %d%s  pMatrix = %p\n", hipGetLastError(), hipGetLastError() != hipSuccess ? " (ERROR)" : "", pMatrix)
	CudaCheckError();
	return true;
}

template<class T>
__global__ void MakeCopyGroupInfoGlobal(CMatrixCanonCheckerGPU<T> **ppCheckers, GroupOrderInfo *orderInfo, int CPU_threadIdx, COrderInfo *pOrderInfo) {
	auto *pEnumInfo = ppCheckers[0]->enumInfo();
	const int iMax = pEnumInfo->GetSize();
	orderInfo += CPU_threadIdx;
	if (!pOrderInfo) {
		if (orderInfo->nOrdersMax < iMax) {
			// Already allocated memory is not enough
			orderInfo->nOrders = -iMax;
			return;
		}

		orderInfo->nOrders = iMax;
		pOrderInfo = orderInfo->pOrderInfo;
	}
	else {
		orderInfo->pOrderInfo = pOrderInfo;
		orderInfo->nOrders = orderInfo->nOrdersMax = iMax;
	}

	for (int i = 0; i < iMax; i++)
		memcpy(pOrderInfo + i, pEnumInfo->GetAt(i), sizeof(COrderInfo));
}

void MakeCopyGroupInfo(CMatrixCanonCheckerGPU<MATRIX_ELEMENT_TYPE> **ppCheckers, GroupOrderInfo *orderInfo, int CPU_threadIdx, hipStream_t stream, COrderInfo *pOrderInfo) {
	MakeCopyGroupInfoGlobal <<<1, 1, 0, stream>>> (ppCheckers, orderInfo, CPU_threadIdx, pOrderInfo);
	CudaCheckError();
}

template<class T>
__global__ void ResetEnumInfoGlobal(CMatrixCanonCheckerGPU<T> **ppCheckers) {
	ppCheckers[0]->enumInfo()->resetEnumInfo();
}

void ResetEnumInfoGlobal(CMatrixCanonCheckerGPU<MATRIX_ELEMENT_TYPE> **ppCheckers, hipStream_t stream) {
	ResetEnumInfoGlobal <<<1, 1, 0, stream>>> (ppCheckers);
	CudaCheckError();
}

template<class T>
__global__ void ReleaseCheckersGlobal(CMatrixCanonCheckerGPU<T> **ppCheckers) {
	delete ppCheckers[blockIdx.x];
}

void ReleaseCheckers(CMatrixCanonCheckerGPU<MATRIX_ELEMENT_TYPE> **ppCheckers, uint numCheckers, hipStream_t stream)
{
	ReleaseCheckersGlobal <<<numCheckers, 1, 0, stream>>> (ppCheckers);
	CudaCheckError();
}

template<class T>
__global__ void TestCanonicity(CMatrixCanonCheckerGPU<T> **ppCheckers, uchar *pCanonFlag, uint *pGroupInfo, uint nObj, int *pMatrFlag, bool noReplicatedBlocks)
{
	auto objIdx = threadIdx.x;
	auto pChecker = ppCheckers[objIdx];
	const CMatrixData<T> * pMatrix = pChecker->matrix();
	pCanonFlag[objIdx] = pChecker->TestCanonicity(pMatrix->rowNumb(), pChecker, t_saveRowPermutations);
	if (pCanonFlag[objIdx]) {
		if (pChecker->groupIsTransitive())
			pCanonFlag[objIdx] |= 0x02;

		pGroupInfo[objIdx] = pChecker->groupOrder();
	}
	
	__syncthreads();
	if (objIdx == 0) {
		CEnumInfo<T> *pEnumInfo = pChecker->enumInfo();
		for (size_t i = 0; i < nObj; i++) {
			if (!pCanonFlag[i])
				continue;

			auto pChecker = ppCheckers[i];
			if (noReplicatedBlocks && pEnumInfo->constructedAllNoReplBlockMatrix())
				pEnumInfo->setNoReplBlockFlag(false);
			else
				pEnumInfo->updateConstrCounters(pMatrFlag[i], pChecker->groupOrder(), pChecker->groupIsTransitive());
		}
	}
}

void TestCanonicity(uint nMatr, CMatrixCanonCheckerGPU<MATRIX_ELEMENT_TYPE> **ppCheckers, uchar *pCanonFlag, uint *pGroupInfo, int *pMatrFlag, hipStream_t stream)
{
	TestCanonicity <<<1, nMatr, 0, stream>>> (ppCheckers, pCanonFlag, pGroupInfo, nMatr, pMatrFlag, false);
	CudaCheckError();
}
#endif
